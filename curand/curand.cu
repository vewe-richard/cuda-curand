#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <ctime>

extern "C"
{
    __declspec(dllexport) void generate_uniform_random(float* hostData, int n);
    __declspec(dllexport) void generate_normal_random(float* hostData, int n, float mean, float stddev);
    __declspec(dllexport) void generate_poisson_random(unsigned int* hostData, int n, double lambda);
}

__global__ void setup_kernel(hiprandState* state, unsigned long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void generate_uniform_kernel(hiprandState* state, float* data, int n) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n) {
        data[id] = hiprand_uniform(&state[id]);
    }
}

__global__ void generate_normal_kernel(hiprandState* state, float* data, int n, float mean, float stddev) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n) {
        data[id] = hiprand_normal(&state[id]) * stddev + mean;
    }
}

__global__ void generate_poisson_kernel(hiprandState* state, unsigned int* data, int n, double lambda) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n) {
        data[id] = hiprand_poisson(&state[id], lambda);
    }
}

// Wrapper functions to be called from C#

void generate_uniform_random(float* hostData, int n) {
    hiprandState* devStates;
    float* devData;

    // Allocate space for cuRAND states on device
    hipMalloc((void**)&devStates, n * sizeof(hiprandState));

    // Allocate space for results on device
    hipMalloc((void**)&devData, n * sizeof(float));

    // Setup the prng states
    setup_kernel << <(n + 255) / 256, 256 >> > (devStates, time(NULL));

    // Generate random numbers
    generate_uniform_kernel << <(n + 255) / 256, 256 >> > (devStates, devData, n);

    // Copy results to host
    hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(devStates);
    hipFree(devData);
}

void generate_normal_random(float* hostData, int n, float mean, float stddev) {
    hiprandState* devStates;
    float* devData;

    hipMalloc((void**)&devStates, n * sizeof(hiprandState));
    hipMalloc((void**)&devData, n * sizeof(float));

    setup_kernel << <(n + 255) / 256, 256 >> > (devStates, time(NULL));
    generate_normal_kernel << <(n + 255) / 256, 256 >> > (devStates, devData, n, mean, stddev);

    hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devStates);
    hipFree(devData);
}

void generate_poisson_random(unsigned int* hostData, int n, double lambda) {
    hiprandState* devStates;
    unsigned int* devData;

    hipMalloc((void**)&devStates, n * sizeof(hiprandState));
    hipMalloc((void**)&devData, n * sizeof(unsigned int));

    setup_kernel << <(n + 255) / 256, 256 >> > (devStates, time(NULL));
    generate_poisson_kernel << <(n + 255) / 256, 256 >> > (devStates, devData, n, lambda);

    hipMemcpy(hostData, devData, n * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(devStates);
    hipFree(devData);
}
