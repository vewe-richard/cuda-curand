#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

extern "C"
{
    __declspec(dllexport) void generate_random_numbers(float* hostData, int n);

}
__global__ void setup_kernel(hiprandState* state, unsigned long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState* state, float* data, int n) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n) {
        data[id] = hiprand_uniform(&state[id]);
    }
}

void generate_random_numbers(float* hostData, int n) {
    hiprandState* devStates;
    float* devData;

    // Allocate space for cuRAND states on device
    hipMalloc((void**)&devStates, n * sizeof(hiprandState));

    // Allocate space for results on device
    hipMalloc((void**)&devData, n * sizeof(float));

    // Setup the prng states
    setup_kernel << <(n + 255) / 256, 256 >> > (devStates, time(NULL));

    // Generate random numbers
    generate_kernel << <(n + 255) / 256, 256 >> > (devStates, devData, n);

    // Copy results to host
    hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(devStates);
    hipFree(devData);
}